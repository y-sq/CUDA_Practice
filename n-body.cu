#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;


//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.
__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double one_over_dis_cube=1.0/pow(sqrt(dis_squared+epsilon_squared),3);
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}



// __constant__ double EPS_SQ;
// __constant__ double DT;
// slower if using constant memory...

template <int THREAD_NUM, bool IS_MULTIPLE>  // int BLOCK_NUM, 
__global__ void N_Body_Simulation(	double* pos_x,double* pos_y,double* pos_z,		////position array
									double* vel_x,double* vel_y,double* vel_z,		////velocity array
									double* new_pos_x,double* new_pos_y,double* new_pos_z,		
									const double* mass,								////mass array
									const int P_N,									////number of particles
									const double DT,								////timestep
									// const double EPS,
									const double EPS_SQ								////epsilon to avoid 0-denominator
								)					
{
	__shared__ double smem_x[THREAD_NUM];
	__shared__ double smem_y[THREAD_NUM];
	__shared__ double smem_z[THREAD_NUM];
	__shared__ double smem_m[THREAD_NUM];

	const int BLOCK_NUM = gridDim.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_s = threadIdx.x;

	double this_x, this_y, this_z;
	double diff_x, diff_y, diff_z;
	double squared, inv_square_root, coef;
	double this_acl_x = 0.0, this_acl_y = 0.0, this_acl_z = 0.0; 

	if (IS_MULTIPLE || idx < P_N) {
		this_x = pos_x[idx];
		this_y = pos_y[idx];
		this_z = pos_z[idx];
	}

	#pragma unroll (4)
	for (int start = 0, k = 0; k < BLOCK_NUM; ++k, start += THREAD_NUM) { 
		int idx_t = start + idx_s; 
		if (IS_MULTIPLE || idx_t < P_N) { 
			smem_x[idx_s] = pos_x[idx_t];
			smem_y[idx_s] = pos_y[idx_t];
			smem_z[idx_s] = pos_z[idx_t];
			smem_m[idx_s] = mass[idx_t];
		}
		__syncthreads();
		#pragma unroll (16)
		for (int i = 0; i < THREAD_NUM; ++i) {
			if (!IS_MULTIPLE && start + i >= P_N) { break; }
			diff_x = smem_x[i] - this_x;
			diff_y = smem_y[i] - this_y;
			diff_z = smem_z[i] - this_z;
			squared = diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;
			inv_square_root = rsqrt(squared + EPS_SQ);
			// inv_square_root = rnorm4d(diff_x, diff_y, diff_z, EPS);
			coef = inv_square_root * inv_square_root * inv_square_root * smem_m[i];
			this_acl_x += coef * diff_x;
			this_acl_y += coef * diff_y;
			this_acl_z += coef * diff_z;
		}
		__syncthreads();
	}
	if (!IS_MULTIPLE && idx >= P_N) { return; }
	double this_vel_x = (vel_x[idx] += this_acl_x * DT);
	double this_vel_y = (vel_y[idx] += this_acl_y * DT);
	double this_vel_z = (vel_z[idx] += this_acl_z * DT);
	new_pos_x[idx] = this_x + this_vel_x * DT;
	new_pos_y[idx] = this_y + this_vel_y * DT;
	new_pos_z[idx] = this_z + this_vel_z * DT;
}


const double dt=0.001;							////time step
const int time_step_num=10;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

const unsigned int grid_size=16;				////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

template <typename T> 
inline __host__ T* copyMemToGpu(T* host_arr, size_t s, bool if_copy = true)
{
	T* dev_arr;
	hipMalloc((void**)&dev_arr, s*sizeof(T));
	if (if_copy) hipMemcpy(dev_arr, host_arr, s*sizeof(T), hipMemcpyHostToDevice);
	return dev_arr;
} 



__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}


	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);


	double* dev_pos_x = copyMemToGpu(pos_x, particle_n);
	double* dev_pos_y = copyMemToGpu(pos_y, particle_n);
	double* dev_pos_z = copyMemToGpu(pos_z, particle_n);
	double* dev_new_pos_x = copyMemToGpu(pos_x, particle_n, false);
	double* dev_new_pos_y = copyMemToGpu(pos_y, particle_n, false);
	double* dev_new_pos_z = copyMemToGpu(pos_z, particle_n, false);
	double* dev_vel_x = copyMemToGpu(vel_x, particle_n);
	double* dev_vel_y = copyMemToGpu(vel_y, particle_n);
	double* dev_vel_z = copyMemToGpu(vel_z, particle_n);
	double* dev_mass  = copyMemToGpu(mass, particle_n);

	// hipMemcpyToSymbol(HIP_SYMBOL(DT), &dt, sizeof(double));
	// hipMemcpyToSymbol(HIP_SYMBOL(EPS_SQ), &epsilon_squared, sizeof(double));

	// TODO: use different thread_num according to particle_n
	const int THREAD_NUM = 128; 
	const int BLOCK_NUM = (particle_n-1) / THREAD_NUM + 1;
	bool is_multiple = ((particle_n % THREAD_NUM) == 0);
	// cout << THREAD_NUM << " " << BLOCK_NUM << " " << is_multiple << endl;

	for(int i=0;i<time_step_num;i++){
		if (is_multiple) {
			N_Body_Simulation<THREAD_NUM, true><<<BLOCK_NUM, THREAD_NUM>>>
			    (  dev_pos_x, dev_pos_y, dev_pos_z,
				   dev_vel_x, dev_vel_y, dev_vel_z,
				   dev_new_pos_x, dev_new_pos_y, dev_new_pos_z,
				   dev_mass,
				   particle_n , dt, //epsilon, 
				   epsilon_squared
				);
		} else {
			N_Body_Simulation<THREAD_NUM, false><<<BLOCK_NUM, THREAD_NUM>>>
				(  dev_pos_x, dev_pos_y, dev_pos_z,
					dev_vel_x, dev_vel_y, dev_vel_z,
					dev_new_pos_x, dev_new_pos_y, dev_new_pos_z,
					dev_mass,
					particle_n, dt, //epsilon, 
					epsilon_squared
				);
		} 
		swap(dev_pos_x, dev_new_pos_x);
		swap(dev_pos_y, dev_new_pos_y);
		swap(dev_pos_z, dev_new_pos_z);
	}

	hipMemcpy(pos_x, dev_pos_x, particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pos_y, dev_pos_y, particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pos_z, dev_pos_z, particle_n*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dev_pos_x);
	hipFree(dev_pos_y);
	hipFree(dev_pos_z);
	hipFree(dev_new_pos_x);
	hipFree(dev_new_pos_y);
	hipFree(dev_new_pos_z);
	hipFree(dev_vel_x);
	hipFree(dev_vel_y);
	hipFree(dev_vel_z);
	hipFree(dev_mass);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);

	printf("GPU runtime: %.4f ms\n",gpu_time);
	cout<<pos_x[particle_n/2]<<" " <<pos_y[particle_n/2]<<" " <<pos_z[particle_n/2]<<endl;
	
	hipEventDestroy(start);
	hipEventDestroy(end);
}

int main()
{
	Test_N_Body_Simulation();
	return 0;
}